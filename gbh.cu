#include "hip/hip_runtime.h"
/*
Original Code From:
Copyright (C) 2006 Pedro Felzenszwalb
Modifications (may have been made) Copyright (C) 2011, 2012
  Chenliang Xu, Jason Corso.

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/
#include <stdlib.h>
#include <stdio.h>
#include <cstdio>
#include <cstdlib>
#include <dirent.h>
#include <stdio.h>
#include <sys/stat.h>
#include <sys/types.h>
#include "image.h"
#include "pnmfile.h"
//#include "segment-image-multi.h"
#include "disjoint-set.h"

#include <iostream> // from segment-image-multi.h
#include <fstream>
#include <vector>
#include <unistd.h>
#include <omp.h>
#include "edges.h"
#include "misc.h"
#include "filter.h"
#include "disjoint-set.h"
#include "segment-graph-multi.h"

#include <algorithm> // from segment-graph-multi.h
#include <cmath>
#include "disjoint-set-s.h"
#include "segment-graph-s.h"

#define num_cores 4 
#define num_edges_s 3088836 

using namespace std;

/* Save Output for oversegmentation*/
/*void generate_output_s(char *path, int num_frame, int width, int height,
                 universe_s *u, int num_vertices, int case_num) {

	int offset = case_num * num_frame; 
        char savepath[1024];
        image<rgb>** output = new image<rgb>*[num_frame];
        rgb* colors = new rgb[num_vertices];
        for (int i = 0; i < num_vertices; i++)
               colors[i] = random_rgb();

        // write out the ppm files.
        int k = 0;
        for (int i = 0; i < num_frame; i++) {
               snprintf(savepath, 1023, "%s/%02d/%05d.ppm", path, k, i + offset + 1);
               output[i] = new image<rgb>(width, height);
               for (int y = 0; y < height; y++) {
                      for (int x = 0; x < width; x++) {
                             int comp = u->find(y * width + x + i * (width * height));
                             imRef(output[i], x, y) = colors[comp];
                      }
               }
               savePPM(output[i], savepath);
        }

	#pragma omp parallel for 
        for (int i = 0; i < num_frame; i++)
               delete output[i];

        delete[] colors;
        delete[] output;
}
*/
// process every image with graph-based segmentation
__global__ void gb(image<float> *smooth_r[], image<float> *smooth_g[], image<float> *smooth_b[],
        int width, int height, float c, edge *edges_remain0[], edge *edges_remain1[], edge *edges_remain2[], edge *edges_remain3[],
        Edge edges0, Edge edges1, Edge edges2, Edge edges3, float *threshold0, float *threshold1,
        float *threshold2, float *threshold3, universe_s *u0, universe_s *u1, universe_s *u2, universe_s *u3) {
  int case_num = blockIdx.x;
  int num_frame = blockDim.x;
  // ----- node number
  int num_vertices = num_frame * width * height;
  switch(case_num) {
    case 0: 
    {
      initialize_edges(edges0, num_frame, width, height, smooth_r, smooth_g, smooth_b, 0);
      //  printf("Finished edge initialization.\n");
      segment_graph_s(num_vertices, num_edges_s, edges0, c, edges_remain0, u0, threshold0);
      //  printf("Finished unit graph segmentation.\n"); 
    }
    break;
    case 1: 
    {
      initialize_edges(edges1, num_frame, width, height, smooth_r, smooth_g, smooth_b, 1);
      //  printf("Finished edge initialization.\n");
      segment_graph_s(num_vertices, num_edges_s, edges1, c, edges_remain1, u1, threshold1);
      //  printf("Finished unit graph segmentation.\n"); 
    }
    break;
    case 2: 
    {
      initialize_edges(edges2, num_frame, width, height, smooth_r, smooth_g, smooth_b, 2);
      //  printf("Finished edge initialization.\n");
      segment_graph_s(num_vertices, num_edges_s, edges2, c, edges_remain2, u2, threshold2);
      //  printf("Finished unit graph segmentation.\n"); 
    }
    break;
    case 3: 
    {
      initialize_edges(edges3, num_frame, width, height, smooth_r, smooth_g, smooth_b, 3);
      //  printf("Finished edge initialization.\n");
      segment_graph_s(num_vertices, num_edges_s, edges3, c, edges_remain3, u3, threshold3);
      //  printf("Finished unit graph segmentation.\n"); 
    }
    break;
    default: break;
  }
//  printf("Finished mess assignment.\n");
}

/* pixel level minimum spanning tree merge */
void segment_graph(universe *mess, vector<edge>* edges_remain, edge *edges, float c, int width, int height, int level,
                image<float> *smooth_r[], image<float> *smooth_g[], image<float> *smooth_b[], int num_frame, char *path) {
	// new vector containing remain edges
	edges_remain->clear();
	printf("Start segmenting graph in parallel.\n");

        int num_vertices = num_frame * width * height;
        int num_bytes = num_edges_s * sizeof(edge); // edge array size
  	int num_bytes_th = num_vertices * sizeof(float); // threshold array size
        int num_bytes_n = num_vertices * sizeof(uni_elt);
	
	int block_size = num_frame;
	int grid_size = num_cores;
	// initialize edges and remained edges array	
	edge **d_edges_remain0 = NULL;  /*edge *d_edges0 = NULL;*/ Edge d_edges0;
	edge **d_edges_remain1 = NULL;  /*edge *d_edges1 = NULL;*/ Edge d_edges1;
	edge **d_edges_remain2 = NULL;  /*edge *d_edges2 = NULL;*/ Edge d_edges2;
	edge **d_edges_remain3 = NULL;  /*edge *d_edges3 = NULL;*/ Edge d_edges3;
	// hipMalloc memory space for edge vectors 
        hipMalloc((void**)&d_edges_remain0, num_bytes);  hipMalloc((void**)&d_edges0, num_bytes);
        hipMalloc((void**)&d_edges_remain1, num_bytes);  hipMalloc((void**)&d_edges1, num_bytes);
        hipMalloc((void**)&d_edges_remain2, num_bytes);  hipMalloc((void**)&d_edges2, num_bytes);
        hipMalloc((void**)&d_edges_remain3, num_bytes);  hipMalloc((void**)&d_edges3, num_bytes);
        // initialize threshold and node array 
   	float *d_th0 = NULL;   	float *d_th1 = NULL;   	float *d_th2 = NULL;   	float *d_th3 = NULL;
        universe_s *d_u0 = new universe_s(num_vertices); universe_s *d_u1 = new universe_s(num_vertices); 
	universe_s *d_u2 = new universe_s(num_vertices); universe_s *d_u3 = new universe_s(num_vertices);
        // allocate memory space for threshold and node array 
        hipMalloc((void**)&d_th0, num_bytes_th); hipMalloc((void**)&d_th1, num_bytes_th);
        hipMalloc((void**)&d_th2, num_bytes_th); hipMalloc((void**)&d_th3, num_bytes_th);
        hipMalloc((void**)&d_u0, num_bytes_n); hipMalloc((void**)&d_u1, num_bytes_n);
        hipMalloc((void**)&d_u2, num_bytes_n); hipMalloc((void**)&d_u3, num_bytes_n);
        
	gb<<<grid_size,block_size>>>(smooth_r, smooth_g, smooth_b, width, height, c, 
             d_edges_remain0, d_edges_remain1, d_edges_remain2, d_edges_remain3,
             d_edges0, d_edges1, d_edges2, d_edges3, d_th0, d_th1, d_th2, d_th3, d_u0, d_u1, d_u2, 
             d_u3);
  	
        universe_s *u0 = new universe_s(num_vertices); universe_s *u1 = new universe_s(num_vertices); 
	universe_s *u2 = new universe_s(num_vertices); universe_s *u3 = new universe_s(num_vertices);
       	hipMemcpy(u0, d_u0, num_bytes_n, hipMemcpyDeviceToHost); hipMemcpy(u1, d_u1, num_bytes_n, hipMemcpyDeviceToHost);
	hipMemcpy(u2, d_u2, num_bytes_n, hipMemcpyDeviceToHost); hipMemcpy(u3, d_u3, num_bytes_n, hipMemcpyDeviceToHost);
	for (int i = 0; i < num_vertices; ++i) 
          mess->set_in_level(i, level, u0->find(i), u0->rank(i), u0->size(i), u0->mst(i)); 
        for (int i = num_vertices; i < 2*num_vertices; ++i) 
          mess->set_in_level(i, level, u1->find(i-num_vertices), u1->rank(i-num_vertices), u1->size(i-num_vertices), u1->mst(i-num_vertices));
        for (int i = 2*num_vertices; i < 3*num_vertices; ++i) 
          mess->set_in_level(i, level, u2->find(i-2*num_vertices), u2->rank(i-2*num_vertices), u2->size(i-2*num_vertices), u2->mst(i-2*num_vertices));
        for (int i = 3*num_vertices; i < 4*num_vertices; ++i) 
          mess->set_in_level(i, level, u3->find(i-3*num_vertices), u3->rank(i-3*num_vertices), u3->size(i-3*num_vertices), u3->mst(i-3*num_vertices));
        
	// output oversegmentation in level 0 of heirarchical system 
/*        generate_output_s(path, num_frame, width, height, u0, num_vertices, 0); 
        generate_output_s(path, num_frame, width, height, u1, num_vertices, 1); 
        generate_output_s(path, num_frame, width, height, u2, num_vertices, 2); 
        generate_output_s(path, num_frame, width, height, u3, num_vertices, 3); 
        generate_output_s(path, num_frame, width, height, u4, num_vertices, 4); 
        generate_output_s(path, num_frame, width, height, u5, num_vertices, 5); 
        generate_output_s(path, num_frame, width, height, u6, num_vertices, 6); 
        generate_output_s(path, num_frame, width, height, u7, num_vertices, 7); 
*/	// transfter edges to edges_remian for first level hierarchical segmentation	
	edge *edges_remain0 = new edge[num_edges_s];  hipMemcpy(edges_remain0, d_edges_remain0, num_bytes, hipMemcpyDeviceToHost);
	edge *edges_remain1 = new edge[num_edges_s];  hipMemcpy(edges_remain1, d_edges_remain1, num_bytes, hipMemcpyDeviceToHost);
	edge *edges_remain2 = new edge[num_edges_s];  hipMemcpy(edges_remain2, d_edges_remain2, num_bytes, hipMemcpyDeviceToHost);
	edge *edges_remain3 = new edge[num_edges_s];  hipMemcpy(edges_remain3, d_edges_remain3, num_bytes, hipMemcpyDeviceToHost);
	// collect remained edges which were not merged in first level graph-based segmentation
        for ( int it = 0; it < num_edges_s; it++ )
          edges_remain->push_back(edges_remain0[it]); 
        for ( int it = 0; it < num_edges_s; it++ )
          edges_remain->push_back(edges_remain1[it]); 
        for ( int it = 0; it < num_edges_s; it++ )
          edges_remain->push_back(edges_remain2[it]); 
        for ( int it = 0; it < num_edges_s; it++ )
          edges_remain->push_back(edges_remain3[it]); 
                
	sort(edges_remain->begin(), edges_remain->end());
	// clear temporary variables
        delete edges_remain0; hipFree(d_edges_remain0); delete edges_remain1; hipFree(d_edges_remain1);
        delete edges_remain2; hipFree(d_edges_remain2); delete edges_remain3; hipFree(d_edges_remain3);
       	hipFree(d_th0); hipFree(d_th1); hipFree(d_th2); hipFree(d_th3);
	hipFree(d_u0); hipFree(d_u1); hipFree(d_u2); hipFree(d_u3);
}

/* Gaussian Smoothing */
void smooth_images(image<rgb> *im[], int num_frame, image<float> *smooth_r[],
		image<float> *smooth_g[], image<float> *smooth_b[], float sigma) {

	int width = im[0]->width();
	int height = im[0]->height();

	image<float>** r = new image<float>*[num_frame];
	image<float>** g = new image<float>*[num_frame];
	image<float>** b = new image<float>*[num_frame];
	#pragma omp parallel for 
	for (int i = 0; i < num_frame; i++) {
		r[i] = new image<float>(width, height);
		g[i] = new image<float>(width, height);
		b[i] = new image<float>(width, height);
	}
	for (int i = 0; i < num_frame; i++) {
		for (int y = 0; y < height; y++) {
			for (int x = 0; x < width; x++) {
				imRef(r[i], x, y) = imRef(im[i], x, y).r;
				imRef(g[i], x, y) = imRef(im[i], x, y).g;
				imRef(b[i], x, y) = imRef(im[i], x, y).b;
			}
		}
	}
	// smooth each color channel
//	#pragma omp parallel for 
	for (int i = 0; i < num_frame; i++) {
		smooth_r[i] = smooth(r[i], sigma);
		smooth_g[i] = smooth(g[i], sigma);
		smooth_b[i] = smooth(b[i], sigma);
	}
	#pragma omp parallel for 
	for (int i = 0; i < num_frame; i++) {
		delete r[i];
		delete g[i];
		delete b[i];
	}
	delete[] r;
	delete[] g;
	delete[] b;
}

/* Save Output */
void generate_output(char *path, int num_frame, int width, int height,
		universe *mess, int num_vertices, int level_total) {

	char savepath[1024];
	image<rgb>** output = new image<rgb>*[num_frame];
	rgb* colors = new rgb[num_vertices];
	for (int i = 0; i < num_vertices; i++)
		colors[i] = random_rgb();

	// write out the ppm files.
	for (int k = 0; k <= level_total; k++) {
		for (int i = 0; i < num_frame; i++) {
			// output 1 higher level than them in GBH and replace k with k+1
			snprintf(savepath, 1023, "%s/%02d/%05d.ppm", path, k, i + 1);
			output[i] = new image<rgb>(width, height);
			for (int y = 0; y < height; y++) {
				for (int x = 0; x < width; x++) {
					int comp = mess->find_in_level(
							y * width + x + i * (width * height), k);
					imRef(output[i], x, y) = colors[comp];
				}
			}
			savePPM(output[i], savepath);
		}
		#pragma omp parallel for 
		for (int i = 0; i < num_frame; i++)
			delete output[i];
	}
	delete[] colors;
	delete[] output;

}

/* main operation steps */
void segment_image(char *path, image<rgb> *im[], int num_frame, float c,
		float c_reg, int min_size, float sigma, int hie_num) {

	// step 1 -- Get information
	int width = im[0]->width();
	int height = im[0]->height();

	// ----- node number
	int num_vertices = num_frame * width * height;
	// ----- edge number
	int num_edges_plane = (width - 1) * (height - 1) * 2 + width * (height - 1)
			+ (width - 1) * height;
	int num_edges_layer = (width - 2) * (height - 2) * 9 + (width - 2) * 2 * 6
			+ (height - 2) * 2 * 6 + 4 * 4;
	int num_edges = num_edges_plane * num_frame
			+ num_edges_layer * (num_frame - 1);

	// ----- hierarchy setup
	vector<vector<edge>*> edges_region;
	edges_region.resize(hie_num + 1);

	// ------------------------------------------------------------------

	// step 2 -- smooth images
	image<float>** smooth_r = new image<float>*[num_frame];
	image<float>** smooth_g = new image<float>*[num_frame];
	image<float>** smooth_b = new image<float>*[num_frame];
	smooth_images(im, num_frame, smooth_r, smooth_g, smooth_b, sigma);
	// ------------------------------------------------------------------

	// step 3 -- build edges
	printf("start build edges\n");
	edge* edges = new edge[num_edges];
	initialize_edges(edges, num_frame, width, height, smooth_r, smooth_g,
			smooth_b, 0);
	printf("end build edges\n");
	// ------------------------------------------------------------------
	printf("The edges' number is %d.\n", num_edges);
	// step 4 -- build nodes
	printf("start build nodes\n");
	universe* mess = new universe(num_frame, width, height, smooth_r, smooth_g,
			smooth_b, hie_num);
	printf("end build nodes\n");
	// ------------------------------------------------------------------

	// step 5 -- over-segmentation
	printf("start over-segmentation\n");
	edges_region[0] = new vector<edge>();
	segment_graph(mess, edges_region[0], edges, c, width, height, 0,
                      smooth_r, smooth_g, smooth_b, num_frame/num_cores, path);

	// optional merging small components
/*	for (int i = 0; i < num_edges; i++) {
		int a = mess->find_in_level(edges[i].a, 0);
		int b = mess->find_in_level(edges[i].b, 0);
		if ((a != b)
				&& ((mess->get_size(a) < min_size)
						|| (mess->get_size(b) < min_size)))
			mess->join(a, b, 0, 0);
	}
	printf("end over-segmentation\n");
	// ------------------------------------------------------------------
*/

	// step 6 -- hierarchical segmentation
	for (int i = 0; i < hie_num; i++) {
		printf("level = %d\n", i);
		// incremental in each hierarchy
		min_size = min_size * 1.2;

		printf("start update\n");
		mess->update(i);
		printf("end update\n");

		printf("start fill edge weight\n");
		fill_edge_weight(*edges_region[i], mess, i);
//		initialize_edges(edges, num_frame, width, height, smooth_r, smooth_g, smooth_b, 0);
		printf("end fill edge weight\n");

		printf("start segment graph region\n");
		edges_region[i + 1] = new vector<edge>();
		segment_graph_region(mess, edges_region[i + 1], edges_region[i], c_reg, i + 1);
		printf("end segment graph region\n");

		printf("start merging min_size\n");
		for (int it = 0; it < (int) edges_region[i]->size(); it++) {
			int a = mess->find_in_level((*edges_region[i])[it].a, i + 1);
			int b = mess->find_in_level((*edges_region[i])[it].b, i + 1);
			if ((a != b)
					&& ((mess->get_size(a) < min_size)
							|| (mess->get_size(b) < min_size)))
				mess->join(a, b, 0, i + 1);
		}
		printf("end merging min_size\n");

		c_reg = c_reg * 1.4;
		delete edges_region[i];
	}
	delete edges_region[hie_num];
	// ------------------------------------------------------------------

	// step 8 -- generate output
	printf("start output\n");
	generate_output(path, num_frame, width, height, mess, num_vertices,
			hie_num);
	printf("end output\n");
	// ------------------------------------------------------------------

	// step 9 -- clear everything
	delete mess;
	delete[] edges;
	#pragma omp parallel for 
	for (int i = 0; i < num_frame; i++) {
		delete smooth_r[i];
		delete smooth_g[i];
		delete smooth_b[i];
	}
	delete[] smooth_r;
	delete[] smooth_g;
	delete[] smooth_b;

}

int main(int argc, char **argv) {
	if (argc != 8) {
		printf("%s c c_reg min sigma hie_num input output\n", argv[0]);
		printf("       c --> value for the threshold function in over-segmentation\n");
		printf("   c_reg --> value for the threshold function in hierarchical region segmentation\n");
		printf("     min --> enforced minimum supervoxel size\n");
		printf("   sigma --> variance of the Gaussian smoothing.\n");
		printf(" hie_num --> desired number of hierarchy levels\n");
		printf("   input --> input path of ppm video frames\n");
		printf("  output --> output path of segmentation results\n");
		return 1;
	}

	// Read Parameters
	float c = atof(argv[1]);
	float c_reg = atof(argv[2]);
	int min_size = atoi(argv[3]);
	float sigma = atof(argv[4]);
	int hie_num = atoi(argv[5]);
	char* input_path = argv[6];
	char* output_path = argv[7];
	if (c <= 0 || c_reg < 0 || min_size < 0 || sigma < 0 || hie_num < 0) {
		fprintf(stderr, "Unable to use the input parameters.");
		return 1;
	}

	// count files in the input directory
	int frame_num = 0;
	struct dirent* pDirent;
	DIR* pDir;
	pDir = opendir(input_path);
	if (pDir != NULL) {
		while ((pDirent = readdir(pDir)) != NULL) {
			int len = strlen(pDirent->d_name);
			if (len >= 4) {
				if (strcmp(".ppm", &(pDirent->d_name[len - 4])) == 0)
					frame_num++;
			}
		}
	}
	if (frame_num == 0) {
		fprintf(stderr, "Unable to find video frames at %s", input_path);
		return 1;
	}
	printf("Total number of frames in fold is %d\n", frame_num);


	// make the output directory
	struct stat st;
	int status = 0;
	char savepath[1024];
  	snprintf(savepath,1023,"%s",output_path);
	if (stat(savepath, &st) != 0) {
		/* Directory does not exist */
		if (mkdir(savepath, S_IRWXU) != 0) {
			status = -1;
		}
	}
	for (int i = 0; i <= hie_num; i++) {
  		snprintf(savepath,1023,"%s/%02d",output_path,i);
		if (stat(savepath, &st) != 0) {
			/* Directory does not exist */
			if (mkdir(savepath, S_IRWXU) != 0) {
				status = -1;
			}
		}
	}
	if (status == -1) {
		fprintf(stderr,"Unable to create the output directories at %s",output_path);
		return 1;
	}


	// Initialize Parameters
	image<rgb>** images = new image<rgb>*[frame_num];
	char filepath[1024];

	// Time Recorder
	time_t Start_t, End_t;
	int time_task;
	Start_t = time(NULL);

	// Read Frames
	for (int i = 0; i < frame_num; i++) {
		snprintf(filepath, 1023, "%s/%05d.ppm", input_path, i + 1);
		images[i] = loadPPM(filepath);
		printf("load --> %s\n", filepath);
	}

	// segmentation
	segment_image(output_path, images, frame_num, c, c_reg, min_size, sigma, hie_num);

	// Time Recorder
	End_t = time(NULL);
	time_task = difftime(End_t, Start_t);
	std::ofstream myfile;
	char timefile[1024];
	snprintf(timefile, 1023, "%s/%s", output_path, "time.txt");
	myfile.open(timefile);
	myfile << time_task << endl;
	myfile.close();

	printf("Congratulations! It's done!\n");
	printf("Time_total = %d seconds\n", time_task);
	return 0;
}

